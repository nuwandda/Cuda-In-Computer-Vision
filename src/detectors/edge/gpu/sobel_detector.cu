#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include <time.h>
#include "Bitmap.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

using namespace std;

#define BLOCK_SIZE 32

__global__ void sobelDetect(unsigned char *in, unsigned char *out, int width, int height) {
    // Kernel to detect edges of the given image. Threads travel on each pixel and use Sobel technique to find edges.
    // If a pixel is an edge then it returns a higher value.
    // Works with gray scaled images.
    //
    // Args:
    //    in (unsigned char): Input image
    //    out (unsigned char): Output image
    //    width (int): Width of the input image
    //    height (int): Height of the input image

    // Set the row and col value for each thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Define derivates
    float dx;
    float dy;

    // Check boundary conditions
    if ((col >= 0 && col < width) && (row >= 0 && row < height)) {
        // Find derivatives
        dx = -in[width * (row - 1) + (col - 1)] + in[width * (row - 1) + (col + 1)]
             - 2 * in[width * row + (col - 1)] + 2 * in[width * (row) + (col + 1)]
             - in[width * (row + 1) + (col - 1)] + in[width * (row + 1) + (col + 1)];

        dy = in[width * (row - 1) + (col - 1)] + 2 * in[width * (row - 1) + (col + 1)] +
             in[width * (row - 1) + (col + 1)]
             - in[width * (row + 1) + (col - 1)] - 2 * in[width * (row + 1) + col] - in[width * (row + 1) + (col + 1)];

        // Get distance
        int value = (int) sqrt(dx * dx + dy * dy);
        // Set the output image values
        out[row * width + col] = (unsigned char) value;
    }
}

int main() {
    // System specifications
    printf("-->\n");
    printf("System Specifications:\n");
    printf("\tAzure NC6\n");
    printf("\tCores: 6\n");
    printf("\tGPU: Tesla K80\n");
    printf("\tMemory: 56 GB\n");
    printf("\tDisk: 380 GB SSD\n");
    printf("-->\n");

    // Create input and output images
    Bitmap *image = new Bitmap();
    Bitmap *outputImage = new Bitmap();

    // Load both images with the same picture
    image->Load("lena.bmp");
    outputImage->Load("lena.bmp");

    // Cuda error and image values
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipError_t status;
    int width = image->Width();
    int height = image->Height();

    // Allocate and copy images
    int size = width * height * sizeof(char);
    unsigned char *deviceInputImage;
    hipMalloc((void **) &deviceInputImage, size);
    status = hipGetLastError();
    if (status != hipSuccess) {
        cout << "Kernel failed for hipMalloc : " << hipGetErrorString(status) << endl;
        return false;
    }

    hipMemcpy(deviceInputImage, image->image, size, hipMemcpyHostToDevice);
    status = hipGetLastError();
    if (status != hipSuccess) {
        cout << "Kernel failed for hipMemcpy hipMemcpyHostToDevice: " << hipGetErrorString(status) << endl;
        hipFree(deviceInputImage);
        return false;
    }

    unsigned char *deviceOutputImage;
    hipMalloc((void **) &deviceOutputImage, size);
    status = hipGetLastError();
    if (status != hipSuccess) {
        cout << "Kernel failed for hipMalloc : " << hipGetErrorString(status) << endl;
        return false;
    }

    // Check for shared memories and call the kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 dimGrid((int) ceil((float) image->Width() / (float) BLOCK_SIZE),
                 (int) ceil((float) image->Height() / (float) BLOCK_SIZE), 1);

    // Check for shared memories and call the kernel
    hipEventRecord(start);
    sobelDetect << < dimGrid, dimBlock >> > (deviceInputImage, deviceOutputImage, width, height);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Save output image to host
    hipMemcpy(outputImage->image, deviceOutputImage, size, hipMemcpyDeviceToHost);
    status = hipGetLastError();
    if (status != hipSuccess) {
        cout << "Kernel failed for hipMemcpy hipMemcpyDeviceToHost: " << hipGetErrorString(status) << endl;
        hipFree(deviceInputImage);
        hipFree(deviceOutputImage);
        return false;
    }
    // Free the memory
    hipFree(deviceInputImage);
    hipFree(deviceOutputImage);

    // Calculate time
    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    printf("Execution time:\n");
    printf("\tSobel edge detection execution time: %f ms\n", time);

    // Save image
    outputImage->Save("lena_shared.bmp");

    return true;
}